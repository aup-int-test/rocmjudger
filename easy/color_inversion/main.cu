#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <iomanip>
#include <hip/hip_runtime.h>

__global__ void invert_kernel(unsigned char* image, int width, int height) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    
    if(idx >= width * height) return;

    int baseidx = idx * 4;

    image[baseidx] = 255 - image[baseidx];
    image[baseidx + 1] = 255 - image[baseidx + 1];
    image[baseidx + 2] = 255 - image[baseidx + 2];
}

void solve(unsigned char* image, int width, int height) {

    unsigned char *d_image;

    hipMalloc(&d_image, width * height * 4 * sizeof(unsigned char));

    hipMemcpy(d_image, image, width * height * 4 * sizeof(unsigned char), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (width * height + threadsPerBlock - 1) / threadsPerBlock;

    invert_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_image, width, height);
    hipDeviceSynchronize();

    hipMemcpy(image, d_image, width * height * 4 * sizeof(unsigned char), hipMemcpyDeviceToHost);

    hipFree(d_image);
}

int main(){
    int width, height;
    std::cin >> width >> height;

    std::vector<unsigned char> image(width * height * 4);
    
    for (int i = 0; i < width * height * 4; ++i){
        int temp;
        std::cin >> temp;
        image[i] = (unsigned char)temp;  
    }

    solve(image.data(), width, height);

    for(int i = 0; i < width * height; ++i){
        for(int j = 0; j < 4; ++j){
            std::cout << (int)image[i * 4 + j] << " ";
        }
        std::cout << std::endl;
    }

    return 0;
}