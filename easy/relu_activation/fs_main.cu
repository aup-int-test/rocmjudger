#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <iomanip>
#include <hip/hip_runtime.h>

#include <fstream>

__global__ void relu_kernel(const float* input, float* output, int N) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if(idx < N){
        output[idx] = fmax(input[idx], 0.0f);
    }
}

void solve(const float* input, float* output, int N) {

    float *d_input, *d_output;

    hipMalloc(&d_input, N * sizeof(float));
    hipMalloc(&d_output, N * sizeof(float));

    hipMemcpy(d_input, input, N * sizeof(float), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    relu_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_output, N);
    hipDeviceSynchronize();

    hipMemcpy(output, d_output, N * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}

int main(int argc, char* argv[]) {
    if (argc != 2) {
        std::cerr << "usage: " << argv[0] << " <input_file>" << std::endl;
        return 1;
    }
    
    std::ifstream input_file;
    std::string filename = argv[1];
    
    input_file.open(filename);
    if (!input_file.is_open()) {
        std::cerr << "fileopen error" << filename << std::endl;
        return 1;
    }
    int N;
    input_file >> N;

    std::vector<float> input(N), output(N);

    for(int i = 0; i < N; ++i) input_file >> input[i];

    input_file.close();

    solve(input.data(), output.data(), N);

    for(int i = 0; i < N; ++i) std::cout << output[i] << " ";
    std::cout << std::endl;
}
