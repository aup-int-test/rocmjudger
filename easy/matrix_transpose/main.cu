#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <iomanip>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 32

__global__ void matrix_transpose_kernel(const float* input, float* output, int rows, int cols) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    if(idx < cols && idy < rows){
        output[idx * rows + idy] = input[idy * cols + idx];
    }
}

void solve(const float* input, float* output, int rows, int cols) {

    float *d_input, *d_output;

    hipMalloc(&d_input, rows * cols * sizeof(float));
    hipMalloc(&d_output, rows * cols * sizeof(float));

    hipMemcpy(d_input, input, rows * cols * sizeof(float), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocksPerGrid((cols + BLOCK_SIZE - 1) / BLOCK_SIZE,
                       (rows + BLOCK_SIZE - 1) / BLOCK_SIZE);

    matrix_transpose_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_output, rows, cols);
    
    hipDeviceSynchronize();

    hipMemcpy(output, d_output, rows * cols * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}

int main(){
    int rows, cols;
    std::cin >> rows >> cols;

    std::vector<float> h_input(rows * cols), h_output(rows * cols);
    
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            std::cin >> h_input[i * cols + j];
        }
    }

    solve(h_input.data(), h_output.data(), rows, cols);

    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            std::cout << h_output[i * cols + j];
            if (j < cols - 1) std::cout << " ";
        }
        std::cout << std::endl;
    }

    return 0;
}