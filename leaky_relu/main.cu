#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <iomanip>
#include <hip/hip_runtime.h>

__global__ void leaky_relu_kernel(const float* input, float* output, int N) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if(idx >= N) return;
    
    // alpha = 0.01, can modify when publish on oj
    output[idx] = input[idx] > 0 ? input[idx] : input[idx] * 0.01;
    
    /*
    if(input[idx] <= 0) output[idx] = input[idx] * 0.01;
    else output[idx] = input[idx];
    */
}

// input, output are device pointers (i.e. pointers to memory on the GPU)
void solve(const float* input, float* output, int N) {

    float *d_input, *d_output;

    hipMalloc(&d_input, N * sizeof(float));
    hipMalloc(&d_output, N * sizeof(float));

    hipMemcpy(d_input, input, N * sizeof(float), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    
    leaky_relu_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_output, N);
    hipDeviceSynchronize();

    hipMemcpy(output, d_output, N * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}


int main(){
    int N;
    std::cin >> N;

    std::vector<float> input(N), output(N);

    for(int i = 0; i < N; ++i) std::cin >> input[i];

    solve(input.data(), output.data(), N);

    std::cout << std::fixed << std::setprecision(3);

    for(int i = 0; i < N; ++i) std::cout << output[i] << " ";
    std::cout << std::endl;
}