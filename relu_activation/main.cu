#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <iomanip>
#include <hip/hip_runtime.h>

__global__ void relu_kernel(const float* input, float* output, int N) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if(idx < N){
        output[idx] = max(input[idx], 0.0f);
    }
}

// input, output are device pointers (i.e. pointers to memory on the GPU)
void solve(const float* input, float* output, int N) {

    float *d_input, *d_output;

    hipMalloc(&d_input, N * sizeof(float));
    hipMalloc(&d_output, N * sizeof(float));

    hipMemcpy(d_input, input, N * sizeof(float), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    relu_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_output, N);
    hipDeviceSynchronize();

    hipMemcpy(output, d_output, N * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}

int main(){
    int N;
    std::cin >> N;

    std::vector<float> input(N), output(N);

    for(int i = 0; i < N; ++i) std::cin >> input[i];

    solve(input.data(), output.data(), N);

    for(int i = 0; i < N; ++i) std::cout << output[i] << " ";
    std::cout << std::endl;
}
