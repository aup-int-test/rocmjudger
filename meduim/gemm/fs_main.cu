#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <iostream>
#include <vector>
#include <iomanip>

#include <fstream>

#define threadperblock 256

__global__ void kernel(const half* A, const half* B, half* C, int M, int N, int K, float alpha, float beta){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int row = idx / N;
    int col = idx % N;
    
    if (row < M && col < N) {

        float sum = 0.0f;
        for (int k = 0; k < K; k++) {
            sum += __half2float(A[row * K + k]) * __half2float(B[k * N + col]);
        }
        
        half old_c = C[row * N + col];
        C[row * N + col] = __hadd(__float2half(alpha * sum), __float2half(beta * __half2float(old_c)));
    }
}

void solve(const half* A, const half* B, half* C, int M, int N, int K, float alpha, float beta) {

    half *d_A, *d_B, *d_C;

    hipMalloc(&d_A, M * K * sizeof(half));
    hipMalloc(&d_B, K * N * sizeof(half));
    hipMalloc(&d_C, M * N * sizeof(half));

    hipMemcpy(d_A, A, M * K * sizeof(half), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, K * N* sizeof(half), hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, M * N* sizeof(half), hipMemcpyHostToDevice);

    int blocks = (M * N + threadperblock - 1) / threadperblock;

    kernel<<<blocks, threadperblock>>>(d_A, d_B, d_C, M, N, K, alpha, beta);

    hipMemcpy(C, d_C, M * N * sizeof(half), hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

int main(int argc, char* argv[]) {
    if (argc != 2) {
        std::cerr << "usage: " << argv[0] << " <input_file>" << std::endl;
        return 1;
    }
    
    std::ifstream input_file;
    std::string filename = argv[1];
    
    input_file.open(filename);
    if (!input_file.is_open()) {
        std::cerr << "fileopen error" << filename << std::endl;
        return 1;
    }

    int M, N, K;
    float alpha, beta;
    
    input_file >> M >> N >> K;
    input_file >> alpha >> beta;

    std::vector<half> A(M * K), B(K * N), C(M * N);

    for(int i = 0; i < M; ++i){
        for(int j = 0; j < K; ++j){
            float temp;
            input_file >> temp;
            A[i * K + j] = __float2half(temp);
        }
    }

    for(int i = 0; i < K; ++i){
        for(int j = 0; j < N; ++j){
            float temp;
            input_file >> temp;
            B[i * N + j] = __float2half(temp);
        }
    }

    for(int i = 0; i < M; ++i){
        for(int j = 0; j < N; ++j){
            float temp;
            input_file >> temp;
            C[i * N + j] = __float2half(temp);
        }
    }

    input_file.close();

    solve(A.data(), B.data(), C.data(), M, N, K, alpha, beta);

    for(int i = 0; i < M; ++i) {
        for(int j = 0; j < N; ++j) {
            float temp;
            C[i * N + j] = __float2half(temp);
            std::cout << temp;
        }
    }
}