#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <iomanip>
#include <hip/hip_runtime.h>

#include <fstream>

#define threadsperblock 1024

__global__ void kernel(const int *input, int *histogram, int N, int num_bins){

    __shared__ int sdata[threadsperblock]; 

    int tidx = threadIdx.x;
    int whichnumbins = blockIdx.x;

    sdata[tidx] = 0;

    for(int i = tidx; i < N; i+= threadsperblock){
        if(input[i] == whichnumbins) sdata[tidx]++;
        
    }

    __syncthreads();

    for (int i = blockDim.x / 2; i > 0; i >>= 1) {
        if (tidx < i) sdata[tidx] += sdata[tidx + i];
        __syncthreads();
    }

    if (tidx == 0) histogram[whichnumbins] = sdata[0]; 
}


extern "C" void solve(const int* input, int* histogram, int N, int num_bins) {
    
    int threads = 256;
    int blocks = (N + threads - 1) / threads;
    int *d_input, *d_histogram;

    hipMalloc(&d_input, N * sizeof(int));
    hipMalloc(&d_histogram, num_bins * sizeof(int));

    hipMemcpy(d_input, input, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_histogram, histogram, num_bins * sizeof(int), hipMemcpyHostToDevice);

    hipOccupancyMaxPotentialBlockSize(&blocks, &threads, kernel, num_bins * sizeof(int));
    
    hipMemset(d_histogram, 0, num_bins * sizeof(int));
    kernel<<<blocks, threads, num_bins * sizeof(int)>>>(d_input, d_histogram, N, num_bins);
    hipDeviceSynchronize();

    hipMemcpy(histogram, d_histogram, num_bins * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_histogram);
}

int main(int argc, char* argv[]) {
    if (argc != 2) {
        std::cerr << "usage: " << argv[0] << " <input_file>" << std::endl;
        return 1;
    }
    
    std::ifstream input_file;
    std::string filename = argv[1];
    
    input_file.open(filename);
    if (!input_file.is_open()) {
        std::cerr << "fileopen error" << filename << std::endl;
        return 1;
    }
    int N, num_bins;
    input_file >> N >> num_bins;

    std::vector<int> input(N), histogram(num_bins);
    
    for(int i = 0; i < N ; ++i) input_file >> input[i];
    
    input_file.close();

    solve(input.data(), histogram.data(), N, num_bins);

    for(int i = 0; i < num_bins ; ++i) std::cout << histogram[i] << " ";
    std::cout << std::endl;
}