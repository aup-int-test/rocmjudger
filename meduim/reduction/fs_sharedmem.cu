#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <iomanip>
#include <hip/hip_runtime.h>

#include <fstream>

#define threadperblock 256

__global__ void reduction(const float* input, float* output, int N){
    __shared__ float sdata[threadperblock];

    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int tidx = threadIdx.x;

    sdata[tidx] = (idx < N) ? input[idx] : 0.0f;

    __syncthreads();

    // Parallel reduction within block
    for (int i = blockDim.x / 2; i > 0; i >>= 1) {
        if (tidx < i) sdata[tidx] += sdata[tidx + i];
        __syncthreads();
    }

    if (tidx == 0) atomicAdd(output, sdata[0]); 
}


extern "C" void solve(const float* input, float* output, int N) {  

    float *d_input, *d_output;

    hipMalloc(&d_input, N * sizeof(float));
    hipMalloc(&d_output, sizeof(float));

    hipMemcpy(d_input, input, N * sizeof(float), hipMemcpyHostToDevice);

    int blockpergrid = (N + threadperblock - 1) / threadperblock;

    //size_t shared_mem_size = threadperblock * sizeof(float);
    reduction<<<blockpergrid, threadperblock>>>(d_input, d_output, N);
    hipDeviceSynchronize();

    hipMemcpy(output, d_output, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}

int main(int argc, char* argv[]){
    std::ifstream infile;
    std::ofstream outfile;
    
    // 決定輸入來源
    if (argc > 1) {
        infile.open(argv[1]);
        if (!infile.is_open()) {
            std::cerr << "Error: Cannot open input file " << argv[1] << std::endl;
            return 1;
        }
    }
    
    // 決定輸出目標
    if (argc > 2) {
        outfile.open(argv[2]);
        if (!outfile.is_open()) {
            std::cerr << "Error: Cannot open output file " << argv[2] << std::endl;
            return 1;
        }
    }
    
    // 選擇輸入流
    std::istream& input_stream = (argc > 1) ? infile : std::cin;
    std::ostream& output_stream = (argc > 2) ? outfile : std::cout;
    
    int N;
    float output;
    
    input_stream >> N;
    std::vector<float> input(N);

    for(int i = 0; i < N; ++i) {
        input_stream >> input[i];
    }

    solve(input.data(), &output, N);

    output_stream << output << std::endl;
    
    // 關閉檔案
    if (infile.is_open()) infile.close();
    if (outfile.is_open()) outfile.close();
    
    return 0;
}