#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <iomanip>
#include <hip/hip_runtime.h>

#include <fstream>

#define threadperblock 256

__global__ void reduction(const int* input, int* output, int N){
    __shared__ int sdata[threadperblock];

    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int tidx = threadIdx.x;

    sdata[tidx] = (idx < N) ? input[idx] : 0.0f;

    __syncthreads();

    // Parallel reduction within block
    for (int i = blockDim.x / 2; i > 0; i >>= 1) {
        if (tidx < i) sdata[tidx] += sdata[tidx + i];
        __syncthreads();
    }

    if (tidx == 0) atomicAdd(output, sdata[0]); 
}


extern "C" void solve(const int* input, int* output, int N) {  

    int *d_input, *d_output;

    hipMalloc(&d_input, N * sizeof(int));
    hipMalloc(&d_output, sizeof(int));

    hipMemcpy(d_input, input, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemset(d_output, 0, sizeof(int));
    
    int blockpergrid = (N + threadperblock - 1) / threadperblock;

    //size_t shared_mem_size = threadperblock * sizeof(int);
    reduction<<<blockpergrid, threadperblock>>>(d_input, d_output, N);
    hipDeviceSynchronize();

    hipMemcpy(output, d_output, sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}

int main(int argc, char* argv[]) {
    if (argc != 2) {
        std::cerr << "usage: " << argv[0] << " <input_file>" << std::endl;
        return 1;
    }
    
    std::ifstream input_file;
    std::string filename = argv[1];
    
    input_file.open(filename);
    if (!input_file.is_open()) {
        std::cerr << "fileopen error" << filename << std::endl;
        return 1;
    }
    int N;
    int output;
    
    input_file >> N;
    std::vector<int> input(N);

    for(int i = 0; i < N; ++i) {
        input_file >> input[i];
    }

    input_file.close();

    solve(input.data(), &output, N);

    std::cout << output << std::endl;
    
    return 0;
}