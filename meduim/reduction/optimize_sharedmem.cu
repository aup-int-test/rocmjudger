#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <iomanip>
#include <hip/hip_runtime.h>

#define threadperblock 256

__global__ void reduction(const int* input, int* output, int N){
    __shared__ int sdata[threadperblock];

    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int tidx = threadIdx.x;

    sdata[tidx] = (idx < N) ? input[idx] : 0.0f;

    // Parallel reduction within block
    for (int i = blockDim.x / 2; i > 0; i >>= 1) {
        if (tidx < i) sdata[tidx] += sdata[tidx + i];
        __syncthreads();
    }

    if (tidx == 0) atomicAdd(output, sdata[0]); 
}


extern "C" void solve(const int* input, int* output, int N) {  

    int *d_input, *d_output;

    hipMalloc(&d_input, N * sizeof(int));
    hipMalloc(&d_output, sizeof(int));

    hipMemcpy(d_input, input, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemset(d_output, 0, sizeof(int));
    
    int blockpergrid = (N + threadperblock - 1) / threadperblock;

    //size_t shared_mem_size = threadperblock * sizeof(int);
    reduction<<<blockpergrid, threadperblock>>>(d_input, d_output, N);
    hipDeviceSynchronize();

    hipMemcpy(output, d_output, sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}

int main(){
    int N;
    int output;
    std::cin >> N;

    std::vector<int> input(N);

    for(int i = 0; i < N; ++i) std::cin >> input[i];

    solve(input.data(), &output, N);

    std::cout << output << std::endl;
}